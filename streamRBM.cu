#include "hip/hip_runtime.h"
#include "cuRBM.h"

__constant__ unsigned nCase;
__constant__ float *data_vis, *data_hid;
__constant__ float *data_v_reco[10];

__device__  float  my_rand(unsigned int *seed) {
	// constants for random no gen.
	unsigned long a = 16807;  		
	unsigned long m = 2147483647;   	// 2^31 - 1
	unsigned long x = (unsigned long) *seed;

	x = (a * x)%m;
	*seed = (unsigned int) x;
 	return ((float)x)/m;
}

__global__ void bias(float *c, float *bb, unsigned offset, unsigned nVH, unsigned sb){
  extern __shared__ float vh_bias[];
  if(blockDim.x * blockIdx.x + threadIdx.x < sb){
    unsigned c_idx = offset + blockDim.x * blockIdx.x + threadIdx.x;
    vh_bias[threadIdx.x] = bb[c_idx];
    for(; c_idx < nCase * nVH; c_idx += nVH)
      c[c_idx] += vh_bias[threadIdx.x];
  }
}

__global__ void biasSampling(float *c, float *bb, unsigned offset, unsigned nVH, unsigned sb){
  extern __shared__ float vh_bias[];
  if(blockDim.x * blockIdx.x + threadIdx.x < sb){
    unsigned c_idx = offset + blockDim.x * blockIdx.x + threadIdx.x;
    vh_bias[threadIdx.x] = bb[c_idx];
    for(; c_idx < nCase * nVH; c_idx += nVH){
      if(my_rand(&c_idx) > 1/(1 + exp(-c[c_idx] - vh_bias[threadIdx.x])))
        c[c_idx] = 0;
      else
        c[c_idx] = 1;
    }
  }
}

__global__ void sumUpVisReco(int ns, unsigned len, float *c){
  for(unsigned i = blockDim.x * blockIdx.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x){
   float s =.0; 
   for(int j = 0; j < ns; ++ j)
     s += *(data_v_reco[j] + i);
   c[i] = s;
  }
}

float *d_weight, *d_a, *d_b;
float *d_data_v, *d_data_h, *d_data_v_reco, * d_data_h_reco;
float *dev_data_v_reco[10];
float *d_vis_data, *d_vis_reco, *d_hid_data, *d_hid_reco, *d_ones;
const float alpha = 1.0f;
const float beta  = .0f;
const float beta_one  = 1.0f;
unsigned currentBatch;
//const float learn_rate  = 0.0001;
const float learn_rate  = 10;
//const float learn_rate_neg  = -0.0001;
const float learn_rate_neg  = -10;

hipblasHandle_t handle;
hiprandGenerator_t gen;
hipStream_t *strm;
hipEvent_t *evt;

void deviceMemoryAlloc();
void deviceMemoryFree();

//unsigned copyMiniBatchToDevice(int idx_batch, hipStream_t *s){
unsigned copyMiniBatchToDevice(int idx_batch){
    /* copy mini batch */
  unsigned nBatch = h_miniBatch > (ninst - idx_batch)? (ninst - idx_batch): h_miniBatch;
  //CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, *s));
  CUBLAS_HANDLE_ERROR(hipblasSetMatrix(nvisible, nBatch, sizeof(float),
                      h_data + idx_batch * nvisible, nvisible, d_data_v, nvisible));
  //HANDLE_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(nCase), &nBatch, sizeof(unsigned), 0,
  //             hipMemcpyHostToDevice, *s));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nCase), &nBatch, sizeof(unsigned), 0,
               hipMemcpyHostToDevice));
  return nBatch;
}

/*
void calcUnits(unsigned nunits, float *dev_data, float *b, int sampled, hipStream_t *s){
  //dim3 g(currentBatch, (nunits- 1)/256 + 1);
  if(sampled)
    addBiasAndSampling<<<(nunits- 1)/256 + 1, 256, 256*sizeof(float), *s>>>(nunits, dev_data, b);
  else
    addBias<<<(nunits- 1)/256 + 1, 256, 256*sizeof(float), *s>>>(nunits, dev_data, b);
  hipError_t ret = hipGetLastError();
  HANDLE_ERROR(ret);
}
*/

void calcViHj(float *dev_v, float *dev_h){
    /* calculate (Hi)data/reco and (Vi)data/reco */
    const float avg_alpha = 1.0/currentBatch;
    hipblasStatus_t ret;
    ret = hipblasSgemv(handle, HIPBLAS_OP_N, nvisible, currentBatch, &avg_alpha, d_data_v, nvisible, d_ones, 1, &beta, dev_v, 1);
    CUBLAS_HANDLE_ERROR(ret);

    ret = hipblasSgemv(handle, HIPBLAS_OP_N, nhidden, currentBatch, &avg_alpha, d_data_h, nhidden, d_ones, 1, &beta, dev_h, 1);
    CUBLAS_HANDLE_ERROR(ret);
}

void calcVHij(unit_t u, unsigned offset, unsigned len){
    /* calculate (Hi)data/reco and (Vi)data/reco */
    const float avg_alpha = 1.0/currentBatch;
    float *vhij, *dev_data_vh;
    int stride;
    switch (u){
      case VISIBLE:
        vhij = d_vis_data + offset;
        dev_data_vh = d_data_v + offset;
        stride = nvisible;
        break;
      case HIDDEN:
        vhij = d_hid_data + offset;
        dev_data_vh = d_data_h + offset;
        stride = nhidden;
        break;
      case VISIBLE_RECO:
        vhij = d_vis_reco + offset;
        dev_data_vh = d_data_v_reco + offset;
        stride = nvisible;
        break;
      case HIDDEN_RECO:
        vhij = d_hid_reco + offset;
        dev_data_vh = d_data_h + offset;
        stride = nhidden;
        break;
      default:
        break;
    }
    hipblasStatus_t ret;
    ret = hipblasSgemv(handle, HIPBLAS_OP_N, len, currentBatch, &avg_alpha, dev_data_vh, stride, d_ones, 1, &beta, vhij, 1);
    CUBLAS_HANDLE_ERROR(ret);
}

void deviceMemoryAlloc(){
  // allocate for visible & hidden data 
  HANDLE_ERROR(hipMalloc((void **)&d_data_v, h_miniBatch * nvisible * sizeof(float)));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data_vis), &d_data_v, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_data_h, h_miniBatch * nhidden * sizeof(float)));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data_hid), &d_data_h, sizeof(float *), 0, hipMemcpyHostToDevice));
  for(int j = 0; j < nStream; ++ j)
    HANDLE_ERROR(hipMalloc((void **)&dev_data_v_reco[j], h_miniBatch * nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data_v_reco), &dev_data_v_reco, nStream * sizeof(float *), 0, hipMemcpyHostToDevice));

  // allocate for vis/hid reconstruction
  HANDLE_ERROR(hipMalloc((void **)&d_data_v_reco, h_miniBatch * nvisible * sizeof(float)));
  //HANDLE_ERROR(hipMalloc((void **)&d_data_h_reco, h_miniBatch * nhidden * sizeof(float)));
  
  // weights 
  HANDLE_ERROR(hipMalloc((void **)&d_weight, nvisible * nStream * streamBatch * sizeof(float)));

  // bias to global memory
  HANDLE_ERROR(hipMalloc((void **)&d_a, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_a, h_a, nvisible * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_b, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_b, h_b, nhidden * sizeof(float), hipMemcpyHostToDevice));
  
  // allocate & copy ones vector
  float *h_ones = (float *)malloc(h_miniBatch * sizeof(float));
  fill_n (h_ones, h_miniBatch, 1);
  HANDLE_ERROR(hipMalloc((void **)&d_ones, h_miniBatch * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_ones, h_ones, h_miniBatch * sizeof(float), hipMemcpyHostToDevice));
  free(h_ones);

  // allocate for Vi Hj
  HANDLE_ERROR(hipMalloc((void **)&d_vis_data, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_vis_reco, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_hid_data, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_hid_reco, nhidden * sizeof(float)));
}

void deviceMemoryFree(){
  HANDLE_ERROR(hipFree(d_data_v));
  HANDLE_ERROR(hipFree(d_data_h));
  HANDLE_ERROR(hipFree(d_weight));
  HANDLE_ERROR(hipFree(d_a));
  HANDLE_ERROR(hipFree(d_b));
  //HANDLE_ERROR(hipFree(d_rand));
  HANDLE_ERROR(hipFree(d_ones));
  HANDLE_ERROR(hipFree(d_vis_data));
  HANDLE_ERROR(hipFree(d_hid_data));
  HANDLE_ERROR(hipFree(d_vis_reco));
  HANDLE_ERROR(hipFree(d_hid_reco));
}

void updateBias(unit_t u, unsigned offset, unsigned len){
    float *d_bias, *d_data, *d_reco;
    if(u == VISIBLE){
      d_bias = d_a + offset;
      d_data = d_vis_data;
      d_reco = d_vis_reco;
    }
    else{
      d_bias = d_b + offset;
      d_data = d_hid_data;
      d_reco = d_vis_reco;
    }

    hipblasStatus_t ret;
    ret = hipblasSaxpy(handle, len, &learn_rate, d_data, 1, d_bias, 1);
    CUBLAS_HANDLE_ERROR(ret);
    ret = hipblasSaxpy(handle, len, &learn_rate_neg, d_reco, 1, d_bias, 1);
    CUBLAS_HANDLE_ERROR(ret);
}

void updateWeight(int offset, int len, float *dev_w){
    hipblasStatus_t ret;
    ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                      nvisible, len, 1, &learn_rate,
	              d_vis_data, nvisible, d_hid_data + offset,
		      len, &beta_one, dev_w, nvisible);
    CUBLAS_HANDLE_ERROR(ret);
    ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                      nvisible, len, 1, &learn_rate_neg,
	              d_vis_reco, nvisible, d_hid_reco + offset,
	              len, &beta_one, dev_w, nvisible);
    CUBLAS_HANDLE_ERROR(ret);
}

void phase1_TillVisibleRecon(int idx_strm){
  unsigned currentStreamBatch;
  float *d_weight_strm = d_weight + idx_strm * streamBatch * nvisible;
  hipblasStatus_t ret;
  for(unsigned streamBatch_start = idx_strm * streamBatch; streamBatch_start < nhidden; streamBatch_start += nStream * streamBatch){
    /* calculate starting position and length */
    if(streamBatch_start + streamBatch > nhidden)
      currentStreamBatch = nhidden - streamBatch_start;
    else
      currentStreamBatch = streamBatch;
      
    /* copy partial weights */
    float *h_weight_strm = h_weight + streamBatch_start * nvisible;
    CUBLAS_HANDLE_ERROR(hipblasSetMatrixAsync(nvisible, currentStreamBatch,
    sizeof(float), h_weight_strm, nvisible, d_weight_strm, nvisible, strm[idx_strm]));

    /* matrix multiplication for hidden units calculation */
    float *d_data_h_strm = d_data_h + streamBatch_start;
    ret = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
          currentStreamBatch, currentBatch, nvisible, &alpha,
          d_weight_strm, nvisible, d_data_v, nvisible, &beta, d_data_h_strm, nhidden);
    CUBLAS_HANDLE_ERROR(ret);

    /* add bias and sampling */
    bias<<<(currentStreamBatch - 1)/256 + 1, 256, 256*sizeof(float), strm[idx_strm]>>>(d_data_h, d_b, streamBatch_start, nhidden, currentStreamBatch);
    hipError_t cuda_ret = hipGetLastError();
    HANDLE_ERROR(cuda_ret);

    /* calculate H_j_data */
    calcVHij(HIDDEN, streamBatch_start, currentStreamBatch);
    
    /* partially reconstruct visible units */
    if(streamBatch_start < nStream * streamBatch)
      ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            nvisible, currentBatch, currentStreamBatch, &alpha,
            d_weight_strm, nvisible, d_data_h_strm, nhidden, &beta, dev_data_v_reco[idx_strm], nvisible);
    else
      ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            nvisible, currentBatch, currentStreamBatch, &alpha,
            d_weight_strm, nvisible, d_data_h_strm, nhidden, &beta_one, dev_data_v_reco[idx_strm], nvisible);

    CUBLAS_HANDLE_ERROR(ret);
  }
}

void phase2(int idx_strm){
  unsigned currentStreamBatch;
  float *d_weight_strm = d_weight + idx_strm * nvisible * streamBatch;
  for(unsigned streamBatch_start = idx_strm * streamBatch; streamBatch_start < nhidden; streamBatch_start += nStream * streamBatch){
    /* calculate starting position and length */
    if(streamBatch_start + streamBatch > nhidden)
      currentStreamBatch = nhidden - streamBatch_start;
    else
      currentStreamBatch = streamBatch;
      
    /* copy partial weights */
    float *h_weight_strm = h_weight + streamBatch_start * nvisible;
    CUBLAS_HANDLE_ERROR(hipblasSetMatrixAsync(nvisible, currentStreamBatch,
    sizeof(float), h_weight_strm, nvisible, d_weight_strm, nvisible, strm[idx_strm]));

    /* matrix multiplication for hidden units calculation */
    float *d_data_h_strm = d_data_h + streamBatch_start;
    hipblasStatus_t ret = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
           currentStreamBatch, currentBatch, nvisible, &alpha,
           d_weight_strm, nvisible, d_data_v_reco, nvisible, &beta, d_data_h_strm, nhidden);
    CUBLAS_HANDLE_ERROR(ret);

    /* add bias and sampling */
    bias<<<(currentStreamBatch - 1)/256 + 1, 256, 256*sizeof(float), strm[idx_strm]>>>(d_data_h, d_b, streamBatch_start, nhidden, currentStreamBatch);
    hipError_t cuda_ret = hipGetLastError();
    HANDLE_ERROR(cuda_ret);

    /* calculate H_j_reco */
    calcVHij(HIDDEN_RECO, streamBatch_start, currentStreamBatch);

    /* update bias for hidden */
    updateBias(HIDDEN, streamBatch_start, currentStreamBatch);

    /* update weights */
    updateWeight(streamBatch_start, currentStreamBatch, d_weight_strm);

    /* copy the new weights back to host */
    CUBLAS_HANDLE_ERROR(hipblasGetMatrixAsync(nvisible, currentStreamBatch,
    sizeof(float), d_weight_strm, nvisible, h_weight_strm, nvisible, strm[idx_strm]));
  }
}

void cublasRunRBM(){
  // data
  float *h_data_h = (float *)malloc(sizeof(float) * nhidden* nvisible);

  float msecTotal = 0.0f;
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, NULL));

  hipblasStatus_t ret;
  ret = hipblasCreate(&handle);
  CUBLAS_HANDLE_ERROR(ret);

  deviceMemoryAlloc();
  
  /* initialize streams and events */
  strm = (hipStream_t *)malloc(nStream * sizeof(hipStream_t));
  evt = (hipEvent_t *)malloc(nStream * sizeof(hipEvent_t));
  for(int j = 0; j < nStream; ++ j){
    HANDLE_ERROR(hipStreamCreate(&strm[j]));
    HANDLE_ERROR(hipEventCreate(&evt[j]));
  }

  /* main loop over all samples by mini-batch */
  for(unsigned i = 0; i < ninst; i += h_miniBatch){
    /* copy mini-batch in default stream */
    CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, NULL));
    currentBatch = copyMiniBatchToDevice(i);

    /* sync for mini-batch copy */
    hipDeviceSynchronize();

    /* calculate V_i_data */
    calcVHij(VISIBLE, 0, nvisible);

    /* concurrent streams */
    for(int j = 0; j < nStream; ++ j){
      CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, strm[j]));
      phase1_TillVisibleRecon(j);
    }

    /* sync for visible recon matrix by all streams and sum up 
       return to default NULL stream, implicit sync */
    int streamUsed;
    if(1.0*nhidden/streamBatch > (nStream - 1))
      streamUsed = nStream;
    else
      streamUsed = (nhidden - 1)/streamBatch + 1;
    sumUpVisReco<<<(currentBatch * nvisible)/256 + 1, 256>>>(streamUsed, currentBatch * nvisible, d_data_v_reco);
    bias<<<(nvisible - 1)/256 + 1, 256, 256*sizeof(float)>>>(d_data_v_reco, d_a, 0, nvisible, nvisible);
    //hipDeviceSynchronize();
    hipError_t cuda_ret = hipGetLastError();
    HANDLE_ERROR(cuda_ret);

    /* calculate V_i_reco */
    CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, NULL));
    calcVHij(VISIBLE_RECO, 0, nvisible);

    /* update bias for visible */
    updateBias(VISIBLE, 0, nvisible);
    hipDeviceSynchronize();

    /* concurrent streams */
    for(int j = 0; j < nStream; ++ j){
      CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, strm[j]));
      phase2(j);
    }
  }
  hipDeviceSynchronize();
  hipblasDestroy(handle);

    //unsigned row = currentBatch;
    //unsigned row = 1;
    //HANDLE_ERROR(hipMemcpy(h_data_h, d_weight, sizeof(float)*row*col, hipMemcpyDeviceToHost));
    //printArray(h_data_h, row, col);
    //printArray(h_weight, row, col);
    //printArray(eigen_data_h, row, col);
    //cout << "sqare norm: " << sqn(h_data_h, eigen_data_h, row * col) << endl;
    /*
    unsigned row = nvisible;
    unsigned col = nhidden;
    cout << "sqare norm: " << sqn(h_weight, eigen_data_h, row * col) << endl;
    */

  HANDLE_ERROR(hipEventRecord(stop, NULL));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
  printf("\tcublas: %.2f msec\n", msecTotal);

  deviceMemoryFree();
  free(h_data_h);
}

