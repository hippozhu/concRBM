#include "hip/hip_runtime.h"
#include "cuRBM.h"

__constant__ int *data, *data_hid;
__constant__ float *data_vis_float, *data_hid_float;
__constant__ unsigned nVis, nHid, nCase, miniBatch, lenVis, lenHid;
__constant__ float *weight, *a, *b;
__constant__ size_t pitch_data, pitch_data_hid, pitch_weight;

__device__ float getData(float* base, int row, int col, size_t pitch){
  return *((float *)((char*)base + row * pitch) + col);
}

__device__ int getData(int* base, int row, int col, size_t pitch){
  return *((int *)((char*)base + row * pitch) + col);
}

__device__ void setData(float* base, int row, int col, size_t pitch, float v){
  *((float *)((char*)base + row * pitch) + col) = v;
}

__global__ void kernel1(){
  __shared__ int ds[32][8];
  __shared__ float ws[256];
  __shared__ float sum[256];
  __shared__ float result[32];

  int tid = threadIdx.x, stride;
  int nActive = miniBatch - blockIdx.x * 32 > 32?32:miniBatch - blockIdx.x * 32;

  int nIter = 0;
  // data prefetching
  /*
  int d;
  float w;
  if(nIter * 256 + tid < nVis)
    w = getData(weight, nIter * 256 + tid, blockIdx.y, pitch_weight);
  if(nIter * 8 + tid % 8 < lenVis && blockIdx.x * 32 + tid/8 < miniBatch)
    d = getData(data, blockIdx.x * 32 + tid/8, nIter * 8 + tid % 8, pitch_data);
  */
  // initialize result
  if(tid < 32)
    result[tid] = .0;


  for(; nIter < (nVis - 1)/256 + 1; ++ nIter){
    // copy data from register to shared memory
    if(nIter * 256 + tid < nVis)
      //ws[tid] = w;
      ws[tid] = getData(weight, nIter * 256 + tid, blockIdx.y, pitch_weight);
    else
      ws[tid] = .0;
    if(nIter * 8 + tid % 8 < lenVis && blockIdx.x * 32 + tid/8 < miniBatch)
      //ds[tid/8][tid%8] = d;
      ds[tid/8][tid%8] = getData(data, blockIdx.x * 32 + tid/8, nIter * 8 + tid % 8, pitch_data);
    else
      ds[tid/8][tid%8] = .0;

    __syncthreads();
    // prefetch next element
    /*
    if((nIter + 1) * 256 + tid < nVis)
      w = getData(weight, (nIter + 1) * 256 + tid, blockIdx.y, pitch_weight);
    if((nIter + 1) * 8 + tid % 8 < lenVis && blockIdx.x * 32 + tid/8 < miniBatch)
      d = getData(data, blockIdx.x * 32 + tid / 8, (nIter + 1) * 8 + tid % 8, pitch_data);
    */
    for(int i = 0; i < nActive; ++i){
      sum[tid] = .0;
      if(nIter * 256 + tid < nVis && (ds[i][tid/32] & (1<<(31-tid%32))))
        sum[tid] = ws[tid];
      stride = 128;
      while(stride > 0){
	__syncthreads();
        if(tid<stride)
	  sum[tid] += sum[tid + stride];
        stride /= 2;
      }
      __syncthreads();
      if(tid==0)
        result[i] += sum[0];
    }
  }

  __syncthreads();
  if(tid < nActive)
    setData(data_hid_float, 32 * blockIdx.x + tid, blockIdx.y, nHid * sizeof(float) , result[tid]);
}

