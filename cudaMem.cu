#include "cuRBM.h"

int *d_data, d_data_hid;
float *d_data_hid_float;
float *d_weight, *d_a, *d_b;
size_t d_pitch_weight, d_pitch_data,  d_pitch_data_hid;

void batchTransfer(unsigned start, unsigned batch_size){
  // Copy data to device coalesced
  int *batch_data = h_data + len * start;
  HANDLE_ERROR(hipMemcpy2D(d_data, d_pitch_data, batch_data, h_pitch_data, width, batch_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nCase), &batch_size, sizeof(unsigned), 0, hipMemcpyHostToDevice));
}

void deviceInit(){
  // basic parametes to constant memory
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(miniBatch), &h_miniBatch, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nVis), &nvisible, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nHid), &nhidden, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  // allocate global memory for data of mini batch 
  HANDLE_ERROR(hipMallocPitch((void **)&d_data, &d_pitch_data, len * sizeof(int), h_miniBatch));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data), &d_data, sizeof(int *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(pitch_data), &d_pitch_data, sizeof(size_t), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(lenVis), &len, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMallocPitch((void **)&d_data_hid, &d_pitch_data_hid, len_hid * sizeof(int), h_miniBatch));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data_hid), &d_data_hid, sizeof(int *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(pitch_data_hid), &d_pitch_data_hid, sizeof(size_t), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(lenHid), &len_hid, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMalloc((void **)&d_data_hid_float, ninst * nhidden * sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data_hid_float), &d_data_hid_float, sizeof(int *), 0, hipMemcpyHostToDevice));

  // weights to global memory
  HANDLE_ERROR(hipMallocPitch((void **)&d_weight, &d_pitch_weight, nhidden * sizeof(float), nvisible));
  HANDLE_ERROR(hipMemcpy2D(d_weight, d_pitch_weight, h_weight, nhidden * sizeof(float),  nhidden * sizeof(float), nvisible, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(weight), &d_weight, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(pitch_weight), &d_pitch_weight, sizeof(size_t), 0, hipMemcpyHostToDevice));
  
  // bias to global memory
  HANDLE_ERROR(hipMalloc((void **)&d_a, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_a, h_a, nvisible * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), &d_a, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_b, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_b, h_b, nhidden * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), &d_b, sizeof(float *), 0, hipMemcpyHostToDevice));
}

