#include "hip/hip_runtime.h"
#include "cuRBM.h"

__constant__ unsigned nCase;
//__constant__ unsigned nVis, nHid, nCase, miniBatch;
//__constant__ float *a, *b, *ones, *vis_data, *vis_reco, *hid_data, *hid_reco;
__device__ unsigned seed;

__device__  float  my_rand() {

	// constants for random no gen.
	unsigned long a = 16807;  		
	unsigned long m = 2147483647;   	// 2^31 - 1
	unsigned long x = (unsigned long) seed;

	x = (a * x)%m;

	seed = (unsigned int) x;

 	return ((float)x)/m;
}

__global__ void addBiasAndSampling(unsigned nVH, float *c, float *bb){
  extern __shared__ float vh_bias[];
  int tid = threadIdx.x;
  if (tid + blockDim.x * blockIdx.x < nVH){
    vh_bias[tid] = bb[tid + blockDim.x * blockIdx.x];
    for(unsigned i = tid + blockDim.x * blockIdx.x; i < nCase * nVH; i += nVH)
      if(my_rand() > 1/(1 + exp(-c[i] - vh_bias[tid])))
        c[i] = 0;
      else
        c[i] = 1;
  }
}

__global__ void addBias(unsigned nVH, float *c, float *bb){
  extern __shared__ float vh_bias[];
  int tid = threadIdx.x;
  if (tid + blockDim.x * blockIdx.x < nVH){
    vh_bias[tid] = bb[tid + blockDim.x * blockIdx.x];
    for(unsigned i = tid + blockDim.x * blockIdx.x; i < nCase * nVH; i += nVH)
      c[i] += vh_bias[tid];
  }
}

float *d_weight, *d_a, *d_b;
float *d_data_v, *d_data_h, *d_rand;
float *d_vis_data, *d_vis_reco, *d_hid_data, *d_hid_reco, *d_ones;
hipblasHandle_t handle;
hiprandGenerator_t gen;
const float alpha = 1.0f;
const float beta  = .0f;
const float beta_one  = 1.0f;
unsigned currentBatch;
const float learn_rate  = 0.0001;
const float learn_rate_neg  = -0.0001;

void deviceMemoryAlloc();
void deviceMemoryFree();

unsigned copyMiniBatchToDevice(int idx_batch){
    /* copy mini batch */
  unsigned currentBatch = h_miniBatch > (ninst - idx_batch)? (ninst - idx_batch): h_miniBatch;
  CUBLAS_HANDLE_ERROR(hipblasSetMatrix(nvisible, currentBatch, sizeof(float),
                      h_data + idx_batch * nvisible, nvisible, d_data_v, nvisible));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nCase), &currentBatch, sizeof(unsigned), 0,
               hipMemcpyHostToDevice));
  return currentBatch;
}

void calcUnits(unsigned nunits, float *dev_data, float *b, int sampled){
  //dim3 g(currentBatch, (nunits- 1)/blockSize + 1);
  if(sampled){
    /* set seed for random number generator, generate random numbers (0, 1] */
    //CURAND_HANDLE_ERROR(hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned) time(NULL)));
    //CURAND_HANDLE_ERROR(hiprandGenerateUniform(gen, d_rand, currentBatch * nunits));
    addBiasAndSampling<<<(nunits- 1)/blockSize + 1, blockSize, blockSize*sizeof(float)>>>(nunits, dev_data, b);
  }
  else
    addBias<<<(nunits- 1)/blockSize + 1, blockSize, blockSize*sizeof(float)>>>(nunits, dev_data, b);
  hipError_t ret = hipGetLastError();
  HANDLE_ERROR(ret);
}

void calcViHj(float *dev_v, float *dev_h){
    /* calculate (Hi)data/reco and (Vi)data/reco */
    const float avg_alpha = 1.0/currentBatch;
    hipblasStatus_t ret;
    ret = hipblasSgemv(handle, HIPBLAS_OP_N, nvisible, currentBatch, &avg_alpha, d_data_v, nvisible, d_ones, 1, &beta, dev_v, 1);
    CUBLAS_HANDLE_ERROR(ret);

    ret = hipblasSgemv(handle, HIPBLAS_OP_N, nhidden, currentBatch, &avg_alpha, d_data_h, nhidden, d_ones, 1, &beta, dev_h, 1);
    CUBLAS_HANDLE_ERROR(ret);
}

void cublasRunRBM(){
  // data
  //unsigned bigger = nvisible < nhidden? nhidden: nvisible;
  float *h_data_h = (float *)malloc(sizeof(float) * 2* nvisible * nvisible);

  float msecTotal = 0.0f;
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, NULL));

  hipblasStatus_t ret;
  ret = hipblasCreate(&handle);
  CUBLAS_HANDLE_ERROR(ret);

  deviceMemoryAlloc();
  
  /* create random generator */
  CURAND_HANDLE_ERROR(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  
  for(unsigned i = 0; i < ninst; i += h_miniBatch){
    currentBatch = copyMiniBatchToDevice(i);

    /* matrix multiplication for hidden units calculation */
    ret = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                      nhidden, currentBatch, nvisible, &alpha,
                      d_weight, nvisible, d_data_v, nvisible, &beta, d_data_h, nhidden);
    CUBLAS_HANDLE_ERROR(ret);
    calcUnits(nhidden, d_data_h, d_b, 1);
    calcViHj(d_vis_data, d_hid_data);
/*
*/
    HANDLE_ERROR(hipMemcpy(h_data_h, d_data_h, sizeof(float)*currentBatch*nhidden, hipMemcpyDeviceToHost));
    //printArray(h_data_h, nhidden, currentBatch);

    /* recontruct visible units */
    ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                      nvisible, currentBatch, nhidden, &alpha,
                      d_weight, nvisible, d_data_h, nhidden, &beta, d_data_v, nvisible);
    CUBLAS_HANDLE_ERROR(ret);
    calcUnits(nvisible, d_data_v, d_a, 1);

    /* recontruct hidden units */
    ret = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                      nhidden, currentBatch, nvisible, &alpha,
                      d_weight, nvisible, d_data_v, nvisible, &beta, d_data_h, nhidden);
    CUBLAS_HANDLE_ERROR(ret);
    calcUnits(nhidden, d_data_h, d_b, 0);
    calcViHj(d_vis_reco, d_hid_reco);

    /* update weight */
    ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                      nvisible, nhidden, 1, &learn_rate,
                      d_vis_data, nvisible, d_hid_data, nhidden, &beta_one, d_weight, nvisible);
    CUBLAS_HANDLE_ERROR(ret);
    ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                      nvisible, nhidden, 1, &learn_rate_neg,
                      d_vis_reco, nvisible, d_hid_reco, nhidden, &beta_one, d_weight, nvisible);
    CUBLAS_HANDLE_ERROR(ret);

    /* update bias */
    ret = hipblasSaxpy(handle, nvisible, &learn_rate, d_vis_data, 1, d_a, 1);
    CUBLAS_HANDLE_ERROR(ret);
    ret = hipblasSaxpy(handle, nvisible, &learn_rate_neg, d_vis_reco, 1, d_a, 1);
    CUBLAS_HANDLE_ERROR(ret);

    ret = hipblasSaxpy(handle, nhidden, &learn_rate, d_hid_data, 1, d_b, 1); 
    CUBLAS_HANDLE_ERROR(ret);
    ret = hipblasSaxpy(handle, nhidden, &learn_rate_neg, d_hid_reco, 1, d_b, 1);
    CUBLAS_HANDLE_ERROR(ret);

/*
    HANDLE_ERROR(hipMemcpy(h_data_h, d_a, sizeof(float)*nvisible, hipMemcpyDeviceToHost));
    printArray(h_data_h, 1, nvisible);
    HANDLE_ERROR(hipMemcpy(h_data_h, d_b, sizeof(float)*nhidden, hipMemcpyDeviceToHost));
    printArray(h_data_h, 1, nhidden);
    //cout << "result:" << h_data_h[0] << " " << h_data_h[1] << " " << h_data_h[nvisible] << endl;
*/
  }
  hipblasDestroy(handle);

  HANDLE_ERROR(hipEventRecord(stop, NULL));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
  printf("\tcublas: %.2f msec\n", msecTotal);

  deviceMemoryFree();
  free(h_data_h);
}

void deviceMemoryFree(){
  HANDLE_ERROR(hipFree(d_data_v));
  HANDLE_ERROR(hipFree(d_data_h));
  HANDLE_ERROR(hipFree(d_weight));
  HANDLE_ERROR(hipFree(d_a));
  HANDLE_ERROR(hipFree(d_b));
  HANDLE_ERROR(hipFree(d_rand));
  HANDLE_ERROR(hipFree(d_ones));
  HANDLE_ERROR(hipFree(d_vis_data));
  HANDLE_ERROR(hipFree(d_hid_data));
  HANDLE_ERROR(hipFree(d_vis_reco));
  HANDLE_ERROR(hipFree(d_hid_reco));
}

void deviceMemoryAlloc(){
  // basic parametes to constant memory
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(miniBatch), &h_miniBatch, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nVis), &nvisible, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nHid), &nhidden, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  // allocate mini batch on device
  HANDLE_ERROR(hipMalloc((void **)&d_data_v, h_miniBatch * nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_data_h, h_miniBatch * nhidden * sizeof(float)));
  
  // weights 
  HANDLE_ERROR(hipMalloc((void **)&d_weight, nhidden * nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_weight, h_weight, nhidden * nvisible * sizeof(float), hipMemcpyHostToDevice));
  
  // bias to global memory
  HANDLE_ERROR(hipMalloc((void **)&d_a, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_a, h_a, nvisible * sizeof(float), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), &d_a, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_b, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_b, h_b, nhidden * sizeof(float), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), &d_b, sizeof(float *), 0, hipMemcpyHostToDevice));
  
  /* allocate memory for random numbers */
  unsigned bigger = nvisible < nhidden? nhidden: nvisible;
  HANDLE_ERROR(hipMalloc((void **)&d_rand, h_miniBatch * bigger * sizeof(float)));

  float *h_ones = (float *)malloc(h_miniBatch * sizeof(float));
  fill_n (h_ones, h_miniBatch, 1);
  HANDLE_ERROR(hipMalloc((void **)&d_ones, h_miniBatch * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_ones, h_ones, h_miniBatch * sizeof(float), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ones), &d_ones, sizeof(float *), 0, hipMemcpyHostToDevice));
  free(h_ones);

  HANDLE_ERROR(hipMalloc((void **)&d_vis_data, nvisible * sizeof(float)));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(vis_data), &d_vh, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_vis_reco, nvisible * sizeof(float)));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(vis_reco), &d_vh, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_hid_data, nhidden * sizeof(float)));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(hid_data), &d_vh, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_hid_reco, nhidden * sizeof(float)));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(hid_reco), &d_vh, sizeof(float *), 0, hipMemcpyHostToDevice));
}

