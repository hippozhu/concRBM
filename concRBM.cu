#include "hip/hip_runtime.h"
#include "cuRBM.h"

__constant__ unsigned nCase;

__device__  float  my_rand(unsigned int *seed) {
	// constants for random no gen.
	unsigned long a = 16807;  		
	unsigned long m = 2147483647;   	// 2^31 - 1
	unsigned long x = (unsigned long) *seed;

	x = (a * x)%m;

	*seed = (unsigned int) x;

 	return ((float)x)/m;
}

__global__ void addBiasAndSampling(unsigned nVH, float *c, float *bb){
  extern __shared__ float vh_bias[];
  int tid = threadIdx.x;
  unsigned seed = blockIdx.x * gridDim.y * blockDim.x + blockIdx.y * blockDim.x + tid;  
  if (tid + blockDim.x * blockIdx.y < nVH){
    vh_bias[tid] = bb[tid + blockDim.x * blockIdx.y];
    for(int i = blockIdx.x * nVH + blockIdx.y * blockDim.x + tid; i < nCase * nVH; i += nVH* gridDim.x)
      //if(rand[i] > 1/(1 + exp(-c[i] - vh_bias[tid])))
      if(my_rand(&seed) > 1/(1 + exp(-c[i] - vh_bias[tid])))
        c[i] = 0;
      else
        c[i] = 1;
  }
}

__global__ void addBias(unsigned nVH, float *c, float *bb){
  extern __shared__ float vh_bias[];
  int tid = threadIdx.x;
  if (tid + blockDim.x * blockIdx.y < nVH){
    vh_bias[tid] = bb[tid + blockDim.x * blockIdx.y];
    for(int i = blockIdx.x * nVH + blockIdx.y * blockDim.x + tid; i < nCase * nVH; i += nVH* gridDim.x)
      c[i] += vh_bias[tid];
  }
}

float *d_weight, *d_a, *d_b;
float *d_data_v, *d_data_h, *d_rand;
float *d_vis_data, *d_vis_reco, *d_hid_data, *d_hid_reco, *d_ones;
hipblasHandle_t handle;
hiprandGenerator_t gen;
const float alpha = 1.0f;
const float beta  = .0f;
const float beta_one  = 1.0f;
unsigned currentBatch;
const float learn_rate  = 0.0001;
const float learn_rate_neg  = -0.0001;

void deviceMemoryAlloc();
void deviceMemoryFree();

unsigned copyMiniBatchToDevice(int idx_batch, hipStream_t *s){
    /* copy mini batch */
  unsigned currentBatch = h_miniBatch > (ninst - idx_batch)? (ninst - idx_batch): h_miniBatch;
  CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, *s));
  CUBLAS_HANDLE_ERROR(hipblasSetMatrix(nvisible, currentBatch, sizeof(float),
                      h_data + idx_batch * nvisible, nvisible, d_data_v, nvisible));
  return currentBatch;
}

void calcUnits(unsigned nunits, float *dev_data, float *b, int sampled, hipStream_t *s){
  dim3 g(currentBatch, (nunits- 1)/256 + 1);
  if(sampled){
    addBiasAndSampling<<<g, 256, 256*sizeof(float), *s>>>(nunits, dev_data, b);
  }
  else
    addBias<<<g, 256, 256*sizeof(float), *s>>>(nunits, dev_data, b);
  hipError_t ret = hipGetLastError();
  HANDLE_ERROR(ret);
}

void calcViHj(float *dev_v, float *dev_h){
    /* calculate (Hi)data/reco and (Vi)data/reco */
    const float avg_alpha = 1.0/currentBatch;
    hipblasStatus_t ret;
    ret = hipblasSgemv(handle, HIPBLAS_OP_N, nvisible, currentBatch, &avg_alpha, d_data_v, nvisible, d_ones, 1, &beta, dev_v, 1);
    CUBLAS_HANDLE_ERROR(ret);

    ret = hipblasSgemv(handle, HIPBLAS_OP_N, nhidden, currentBatch, &avg_alpha, d_data_h, nhidden, d_ones, 1, &beta, dev_h, 1);
    CUBLAS_HANDLE_ERROR(ret);
}

void deviceMemoryFree(){
  HANDLE_ERROR(hipFree(d_data_v));
  HANDLE_ERROR(hipFree(d_data_h));
  HANDLE_ERROR(hipFree(d_weight));
  HANDLE_ERROR(hipFree(d_a));
  HANDLE_ERROR(hipFree(d_b));
  HANDLE_ERROR(hipFree(d_rand));
  HANDLE_ERROR(hipFree(d_ones));
  HANDLE_ERROR(hipFree(d_vis_data));
  HANDLE_ERROR(hipFree(d_hid_data));
  HANDLE_ERROR(hipFree(d_vis_reco));
  HANDLE_ERROR(hipFree(d_hid_reco));
}

void deviceMemoryAlloc(){
  // allocate mini batch on device
  HANDLE_ERROR(hipMalloc((void **)&d_data_v, h_miniBatch * nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_data_h, h_miniBatch * nhidden * sizeof(float)));
  
  // weights 
  HANDLE_ERROR(hipMalloc((void **)&d_weight, nvisible * nStream * streamBatch * sizeof(float)));

  // bias to global memory
  HANDLE_ERROR(hipMalloc((void **)&d_a, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_a, h_a, nvisible * sizeof(float), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), &d_a, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_b, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_b, h_b, nhidden * sizeof(float), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), &d_b, sizeof(float *), 0, hipMemcpyHostToDevice));
  
  /* allocate memory for random numbers */
  unsigned bigger = nvisible < nhidden? nhidden: nvisible;
  HANDLE_ERROR(hipMalloc((void **)&d_rand, h_miniBatch * bigger * sizeof(float)));

  float *h_ones = (float *)malloc(h_miniBatch * sizeof(float));
  fill_n (h_ones, h_miniBatch, 1);
  HANDLE_ERROR(hipMalloc((void **)&d_ones, h_miniBatch * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_ones, h_ones, h_miniBatch * sizeof(float), hipMemcpyHostToDevice));
  free(h_ones);

  HANDLE_ERROR(hipMalloc((void **)&d_vis_data, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_vis_reco, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_hid_data, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_hid_reco, nhidden * sizeof(float)));
}

void calcHidden(hipStream_t strm[]){
  for(unsigned k = 0; k < nhidden; k += nStream * streamBatch){

      unsigned currentStreamBatch = streamBatch; 
      unsigned streamBatch_start = k * nStream * streamBatch; 
      for(int j = 0; j < nStream; ++ j){
	if(streamBatch_start > nhidden)
	  break;

        CUBLAS_HANDLE_ERROR(hipblasSetStream(handle, strm[j]));

	if(streamBatch_start + streamBatch > nhidden)
	  currentStreamBatch = nhidden - streamBatch_start;

        // copy partial weights 
	float * h_weight_sb = h_weight + streamBatch_start * nvisible;
	float * d_weight_sb = d_weight + j * nvisible;
        CUBLAS_HANDLE_ERROR(hipblasSetMatrix(nvisible, currentStreamBatch, sizeof(float),
            h_weight_sb, nvisible, d_weight_sb, nvisible));

        /* matrix multiplication for hidden units calculation */
	float * d_data_h_sb = d_data_h + streamBatch_start;
        hipblasStatus_t ret = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                      currentStreamBatch, currentBatch, nvisible, &alpha,
                      d_weight_sb, nvisible, d_data_v, nvisible, &beta, d_data_h_sb, nhidden);
        CUBLAS_HANDLE_ERROR(ret);
        calcUnits(nhidden, d_data_h, d_b, 0, &strm[j]);
        calcViHj(d_vis_data, d_hid_data);
	streamBatch_start += streamBatch;
      }

      /* recontruct visible units */
      for(int j = 0; j < nStream; ++ j){
	if(j>0)
	  HANDLE_ERROR(hipStreamWaitEvent());
        ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                      nvisible, currentBatch, nhidden, &alpha,
                      d_weight, nvisible, d_data_h, nhidden, &beta, d_data_v, nvisible);
        CUBLAS_HANDLE_ERROR(ret);
      }
  }
}


void cublasRunRBM(){
  // data
  //unsigned bigger = nvisible < nhidden? nhidden: nvisible;
  float *h_data_h = (float *)malloc(sizeof(float) * nhidden* nvisible);

  float msecTotal = 0.0f;
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, NULL));

  hipblasStatus_t ret;
  ret = hipblasCreate(&handle);
  CUBLAS_HANDLE_ERROR(ret);

  deviceMemoryAlloc();
  
  /* initialize streams and events */
  hipStream_t strm[nStream];
  hipEvent_t evt[nStream];
  for(int j = 0; j < nStream; ++ j){
    HANDLE_ERROR(hipStreamCreate(&strm[j]));
    HANDLE_ERROR(hipEventCreate(&evt[j]));
  }

  /* main loop over all samples by mini-batch */
  for(unsigned i = 0; i < ninst; i += h_miniBatch){
    currentBatch = copyMiniBatchToDevice(i, &strm[0]);
    HANDLE_ERROR(hipEventRecord(evt[0], strm[0]));

    /* sync for mini-batch copy */
    for(int j = 1; j < nStream; ++ j)
      HANDLE_ERROR(hipStreamWaitEvent(strm[j], evt[0], 0));

    /* first calculation for hidden */
    calcHidden(strm);
    hipDeviceSynchronize();

    calcVisible(strm);
    hipDeviceSynchronize();
  }
  hipblasDestroy(handle);

  HANDLE_ERROR(hipEventRecord(stop, NULL));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
  printf("\tcublas: %.2f msec\n", msecTotal);

  deviceMemoryFree();
  free(h_data_h);
}
